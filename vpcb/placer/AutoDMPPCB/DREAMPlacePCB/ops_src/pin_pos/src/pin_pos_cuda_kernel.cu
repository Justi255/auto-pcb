#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

/// @brief Compute pin position from node position 
template <typename T, typename K>
__global__ void computePinPos(
	const T* x, const T* y,
	const T* pin_offset_x,
	const T* pin_offset_y,
	const K* pin2node_map,
	const int num_pins,
	T* pin_x, T* pin_y
	)
{
	// 一个块x维度有blockDim.x个线程，第blockIdx.x个x维度，偏移量为threadIdx.x
	// 确定线程索引
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_pins)
	{
		int node_id = pin2node_map[i];
		pin_x[i] = pin_offset_x[i] + x[node_id];
		pin_y[i] = pin_offset_y[i] + y[node_id];
	}
}

template <typename T>
int computePinPosCudaLauncher(
	const T* x, const T* y,
	const T* pin_offset_x,
	const T* pin_offset_y,
	const long* pin2node_map,
	const int* flat_node2pin_map,
	const int* flat_node2pin_start_map,
	int num_pins,
	T* pin_x, T* pin_y
    )
{
	int thread_count = 512;

	
	// 在CUDA编程中，<<<...>>> 运算符是用于启动 GPU 核函数的语法。这个运算符的内部包含两个参数，用于指定线程组织的方式。
	// 语法为：<<<num_blocks, threads_per_block>>>
	// num_blocks: 表示在 GPU 上启动的线程块数目。
	// threads_per_block: 表示每个线程块中的线程数量。
	computePinPos<<<(num_pins+thread_count-1) / thread_count, thread_count>>>(x, y, pin_offset_x, pin_offset_y, pin2node_map, num_pins, pin_x, pin_y);

    return 0;
}

/// @brief Compute pin position from node position 
template <typename T>
__global__ void computeNodeGrad(
	const T* grad_out_x,
	const T* grad_out_y,
	const int* flat_node2pin_map,
    const int* flat_node2pin_start_map, 
    const int num_nodes, 
	T* grad_x,
	T* grad_y
	)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nodes)
	{
        T& gx = grad_x[i];
        T& gy = grad_y[i];
        gx = 0; 
        gy = 0; 
        for (int j = flat_node2pin_start_map[i]; j < flat_node2pin_start_map[i+1]; ++j)
        {
            int pin_id = flat_node2pin_map[j]; 
			// grad_out_x 和 grad_out_y 的值是通过神经网络训练过程中的反向传播计算得到的，表示了每个 pin 对输出的影响
            gx += grad_out_x[pin_id]; 
            gy += grad_out_y[pin_id]; 
        }
	}
}

template <typename T>
int computePinPosGradCudaLauncher(
	const T* grad_out_x, const T* grad_out_y,
	const T* x, const T* y,
	const T* pin_offset_x,
	const T* pin_offset_y,
	const long* pin2node_map,
	const int* flat_node2pin_map,
	const int* flat_node2pin_start_map,
	int num_nodes,
	int num_pins,
	T* grad_x, T* grad_y
    )
{
    int thread_count = 512;

    computeNodeGrad<<<(num_nodes + thread_count - 1) / thread_count, thread_count>>>(
            grad_out_x, 
            grad_out_y, 
            flat_node2pin_map, 
            flat_node2pin_start_map, 
            num_nodes, 
            grad_x, 
            grad_y
            );

    return 0;	
}


#define REGISTER_KERNEL_LAUNCHER(T) \
    template int computePinPosCudaLauncher<T>(\
    	    const T* x, const T* y, \
    	    const T* pin_offset_x, \
	        const T* pin_offset_y, \
	        const long* pin2node_map, \
	        const int* flat_node2pin_map, \
	        const int* flat_node2pin_start_map, \
	        int num_pins, \
	        T* pin_x, T* pin_y \
            ); \
    \
    template int computePinPosGradCudaLauncher<T>(\
        	const T* grad_out_x, const T* grad_out_y, \
	        const T* x, const T* y, \
	        const T* pin_offset_x, \
	        const T* pin_offset_y, \
	        const long* pin2node_map, \
	        const int* flat_node2pin_map, \
	        const int* flat_node2pin_start_map, \
	        int num_nodes, \
	        int num_pins, \
	        T* grad_x, T* grad_y \
            ); 

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
